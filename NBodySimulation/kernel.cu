#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""
#include "glad/glad.h"
#include "glfw/glfw3.h"
#include <iostream>
#include <stdio.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void framebuffer_size_callback(GLFWwindow *window, int width, int height);


__global__ void addKernel(int *c, const int *a, const int *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

int main()
{
	//Make CUDA and OpenGL working togheter... done!
	printf("Starting...\n\n");
	printf(
		" CUDA Device Query (Runtime API) version (CUDART static linking)\n\n");

	int deviceCount = 0;
	hipError_t error_id = hipGetDeviceCount(&deviceCount);

	if (error_id != hipSuccess) {
		printf("hipGetDeviceCount returned %d\n-> %s\n",
			static_cast<int>(error_id), hipGetErrorString(error_id));
		printf("Result = FAIL\n");
		exit(EXIT_FAILURE);
	}

	// This function call returns 0 if there are no CUDA capable devices.
	if (deviceCount == 0) {
		printf("There are no available device(s) that support CUDA\n");
	}
	else {
		printf("Detected %d CUDA Capable device(s)\n", deviceCount);
	}

	int dev, driverVersion = 0, runtimeVersion = 0;

	for (dev = 0; dev < deviceCount; ++dev) {
		hipSetDevice(dev);
		hipDeviceProp_t deviceProp;
		hipGetDeviceProperties(&deviceProp, dev);

		printf("\nDevice %d: \"%s\"\n", dev, deviceProp.name);

		// Console log
		hipDriverGetVersion(&driverVersion);
		hipRuntimeGetVersion(&runtimeVersion);
		printf("  CUDA Driver Version / Runtime Version          %d.%d / %d.%d\n",
			driverVersion / 1000, (driverVersion % 100) / 10,
			runtimeVersion / 1000, (runtimeVersion % 100) / 10);
		printf("  CUDA Capability Major/Minor version number:    %d.%d\n",
			deviceProp.major, deviceProp.minor);
	}

	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 3);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 3);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	GLFWwindow *window = glfwCreateWindow(800, 600, "LearnOpenGL", NULL, NULL);
	if (window == NULL) {
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize GLAD" << std::endl;
	}
	glViewport(0, 0, 800, 600);
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	while (!glfwWindowShouldClose(window)) {
		glfwSwapBuffers(window);
		glfwPollEvents();
	}
	glfwTerminate();
	return 0;
}

void framebuffer_size_callback(GLFWwindow * window, int width, int height)
{
	glViewport(0, 0, width, height);
}
