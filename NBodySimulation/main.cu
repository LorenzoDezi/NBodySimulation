#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include "glad/glad.h"
#include "Model.h"
#include "Shader.h"
#include "Mesh.h"
#include "Camera.h"
#define STB_IMAGE_IMPLEMENTATION
#include "stb_image/stb_image.h"
#include "glfw/glfw3.h"
#include <iostream>
#include <glm/gtc/type_ptr.hpp>
#include <stdio.h>

#define N 1000
#define RADIUS 100
#define MASS_SEED 100


void framebuffer_size_callback(GLFWwindow *window, int width, int height);
void mouse_callback(GLFWwindow* w, double xpos, double ypos);
void processInput(GLFWwindow *w);

float lastFrame = 0.0f;
float deltaTime = 0.0f;
bool firstMouse = true;
float lastX = 0.0f;
float lastY = 0.0f;
Camera camera(glm::vec3(0.f, 0.f, 80.f), glm::vec3(0.f, 1.0f, 0.0f));

int main()
{
	glfwInit();
	glfwWindowHint(GLFW_CONTEXT_VERSION_MAJOR, 4);
	glfwWindowHint(GLFW_CONTEXT_VERSION_MINOR, 4);
	glfwWindowHint(GLFW_OPENGL_PROFILE, GLFW_OPENGL_CORE_PROFILE);
	GLFWwindow *window = glfwCreateWindow(800, 600, "LearnOpenGL", NULL, NULL);
	if (window == NULL) {
		std::cout << "Failed to create GLFW window" << std::endl;
		glfwTerminate();
		return -1;
	}
	glfwMakeContextCurrent(window);
	if (!gladLoadGLLoader((GLADloadproc)glfwGetProcAddress)) {
		std::cout << "Failed to initialize GLAD" << std::endl;
	}
	glViewport(0, 0, 800, 600);
	glfwSetInputMode(window, GLFW_CURSOR, GLFW_CURSOR_DISABLED);
	glfwSetCursorPosCallback(window, mouse_callback);
	glEnable(GL_DEPTH_TEST);

	Shader shader("vertex_planet.glsl", "fragment.glsl");
	Shader instancingShader("vertex.glsl", "fragment.glsl");
	
	//Model loading
	Model planetModel("assets/planet.obj");
	Model rockModel("assets/rock.obj");

	// generate a large list of semi-random model transformation matrices
	glm::mat4* modelMatrices;
	modelMatrices = new glm::mat4[N];
	srand(glfwGetTime()); // initialize random seed	
	for (unsigned int i = 0; i < N; i++)
	{
		glm::mat4 model = glm::mat4(1.0f);
		
		//Position: random point inside a sphere of radius RADIUS
		float x, y, z;
		float radius = -RADIUS + (rand() % RADIUS); //random radius
		float theta = rand() % 360; //random angle on xz plane
		float gamma = rand() % 360; //random angle on yz/yx plane
		//Random point inside sphere
		x = radius * cos(theta) * cos(gamma);
		y = radius * sin(gamma);
		z = radius * sin(theta) * cos(gamma);
		//Model matrix
		model = glm::translate(model, glm::vec3(x, y, z));

		// Scale: scale depending on mass
		float mass = (rand() % MASS_SEED) / 100.0f + 0.5f;
		//TODO: save mass on data structure
		model = glm::scale(model, glm::vec3(mass));

		// Rotation: add random rotation around a randomly picked rotation axis vector
		float rotAngle = (rand() % 360);
		model = glm::rotate(model, rotAngle, glm::vec3(0.4f, 0.6f, 0.8f));
		
		modelMatrices[i] = model;
	}

	// vertex Buffer Object
	unsigned int buffer;
	glGenBuffers(1, &buffer);
	glBindBuffer(GL_ARRAY_BUFFER, buffer);
	glBufferData(GL_ARRAY_BUFFER, N * sizeof(glm::mat4), &modelMatrices[0], GL_STATIC_DRAW);
	//TODO Bind also to cuda. Consider passing only positions, and then to the model matrix calculation inside 
	//the shaders. You need position and acceleration, then each shader will calculate the model matrix based on that
	std::vector<int> VAOs = rockModel.GetVAOs();
	for (unsigned int i = 0; i < VAOs.size(); i++)
	{
		unsigned int VAO = VAOs[i];
		glBindVertexArray(VAO);
		// vertex Attributes
		GLsizei vec4Size = sizeof(glm::vec4);
		glEnableVertexAttribArray(3);
		glVertexAttribPointer(3, 4, GL_FLOAT, GL_FALSE, 4 * vec4Size, (void*)0);
		glEnableVertexAttribArray(4);
		glVertexAttribPointer(4, 4, GL_FLOAT, GL_FALSE, 4 * vec4Size, (void*)(vec4Size));
		glEnableVertexAttribArray(5);
		glVertexAttribPointer(5, 4, GL_FLOAT, GL_FALSE, 4 * vec4Size, (void*)(2 * vec4Size));
		glEnableVertexAttribArray(6);
		glVertexAttribPointer(6, 4, GL_FLOAT, GL_FALSE, 4 * vec4Size, (void*)(3 * vec4Size));

		glVertexAttribDivisor(3, 1);
		glVertexAttribDivisor(4, 1);
		glVertexAttribDivisor(5, 1);
		glVertexAttribDivisor(6, 1);

		glBindVertexArray(0);
	}
	
	glfwSetFramebufferSizeCallback(window, framebuffer_size_callback);
	while (!glfwWindowShouldClose(window)) {
		//Delta-time per frame logic
		float currentFrame = glfwGetTime();
		deltaTime = currentFrame - lastFrame;
		lastFrame = currentFrame;
		processInput(window);

		glClearColor(0.1f, 0.1f, 0.1f, 1.0f);
		glClear(GL_COLOR_BUFFER_BIT | GL_DEPTH_BUFFER_BIT);

		//camera transformations setup
		glm::mat4 projection = glm::perspective(glm::radians(45.0f), 1280.f / 720.f, 0.1f, 1000.0f);
		glm::mat4 view = camera.GetViewMatrix();;
		//Shader camera transformation pass
		shader.use();
		shader.setMat4Float("projection", glm::value_ptr(projection));
		shader.setMat4Float("view", glm::value_ptr(view));
		instancingShader.use();
		instancingShader.setMat4Float("projection", glm::value_ptr(projection));
		instancingShader.setMat4Float("view", glm::value_ptr(view));

		//draw objects
		instancingShader.use();
		rockModel.DrawInstanced(instancingShader, N);

		glfwSwapBuffers(window);
		glfwPollEvents();
		
	}
	glfwTerminate();
	return 0;
}

void framebuffer_size_callback(GLFWwindow * window, int width, int height)
{
	glViewport(0, 0, width, height);
}

void mouse_callback(GLFWwindow * w, double xpos, double ypos) {
	if (firstMouse)
	{
		lastX = xpos;
		lastY = ypos;
		firstMouse = false;
	}

	float xoffset = xpos - lastX;
	float yoffset = lastY - ypos;
	lastX = xpos;
	lastY = ypos;
	camera.ProcessMouseMovement(xoffset, yoffset);
}

void processInput(GLFWwindow *w) {
	if (glfwGetKey(w, GLFW_KEY_ESCAPE)) {
		glfwSetWindowShouldClose(w, true);
	}
	if (glfwGetKey(w, GLFW_KEY_UP)) {
		camera.ProcessKeyboard(FORWARD, deltaTime);
	}
	if (glfwGetKey(w, GLFW_KEY_DOWN)) {
		camera.ProcessKeyboard(BACKWARD, deltaTime);
	}
	if (glfwGetKey(w, GLFW_KEY_LEFT)) {
		camera.ProcessKeyboard(LEFT, deltaTime);
	}
	if (glfwGetKey(w, GLFW_KEY_RIGHT)) {
		camera.ProcessKeyboard(RIGHT, deltaTime);
	}
}
